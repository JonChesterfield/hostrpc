#include <hip/hip_runtime.h>

// Intent is to use the cuda calls initially, then transform to clang intrinsics
// and move into platform.hpp

#define DEVICE __device__ __attribute__((always_inline))

#define WARPSIZE 32

namespace platform
{
DEVICE uint32_t get_lane_id(void)
{
  return __nvvm_read_ptx_sreg_tid_x() /*threadIdx.x*/ & (WARPSIZE - 1);
}

#ifndef CUDA_VERSION
#error "Require CUDA_VERSION definition"
#endif

// Something strange here. CUDA_VERSION picks activemask, but
// sm_50 maps onto ptx 4.0 by default which doesn't support that
// Compiling with cuda overrides to ptx 6.3, passing Xclang to match.

namespace detail
{
static DEVICE uint32_t ballot()
{
#if CUDA_VERSION >= 9000
  return __activemask();
#else
  return __ballot(1);
#endif
}

DEVICE int32_t __impl_shfl_down_sync(int32_t var, uint32_t laneDelta)
{
  return __shfl_down_sync(UINT32_MAX, var, laneDelta, WARPSIZE);
}

DEVICE
void(debug)(const char *file, unsigned int line, const char *func,
            unsigned long long value)
{
  uint32_t lane_id = get_lane_id();
  printf("Debug[%u] %s: %s: %d: %llu\n", lane_id, file, func, line, value);
}

DEVICE
void assert_fail(const char *str, const char *file, unsigned int line,
                 const char *func)
{
  uint32_t lane_id = get_lane_id();
  asm("// Assert fail " ::"r"(line), "r"(str));
  printf("Assert fail[%u]: %s (%s: %s)\n", lane_id, str, file, func);
  __builtin_trap();
}

}  // namespace detail

static DEVICE uint32_t get_master_lane_id(void)
{
  // TODO: openmp deviceRTL uses:
  // return (blockDim.x - 1) & ~(WARPSIZE - 1);
  uint32_t activemask = detail::ballot();
  uint32_t lowest_active = __builtin_ffs(activemask) - 1;
  return lowest_active;
}

DEVICE bool is_master_lane() { return get_lane_id() == get_master_lane_id(); }

DEVICE uint32_t broadcast_master(uint32_t x)
{
  uint32_t master_id = get_master_lane_id();
  // __nvvm_shfl_sync_idx_i32(UINT32_MAX, x, master_id, 31)
#if CUDA_VERSION >= 9000
  // Use activemask?
  return __shfl_sync(UINT32_MAX, x, master_id);
#else
  // This may be UB if some lanes are inactive
  return __shfl(x, master_id);
#endif
}

DEVICE uint32_t all_true(uint32_t x) { return __nvvm_vote_all(x); }

// TODO: Check the differences between threadfence, threadfence_block,
// threadfence_system
DEVICE void fence_acquire() { __threadfence_system(); }
DEVICE void fence_release() { __threadfence_system(); }

namespace detail
{
// Might be able to use volatile _Atomic as the top level type if
// opencl load/store is compiled correctly when called from cuda

#define HOSTRPC_STAMP_MEMORY(TYPE)                                            \
  DEVICE TYPE atomic_load_relaxed(volatile _Atomic(TYPE) const *addr)         \
  {                                                                           \
    return __opencl_atomic_load(addr, __ATOMIC_RELAXED,                       \
                                __OPENCL_MEMORY_SCOPE_ALL_SVM_DEVICES);       \
  }                                                                           \
                                                                              \
  DEVICE void atomic_store_relaxed(volatile _Atomic(TYPE) * addr, TYPE value) \
  {                                                                           \
    return __opencl_atomic_store(addr, value, __ATOMIC_RELAXED,               \
                                 __OPENCL_MEMORY_SCOPE_ALL_SVM_DEVICES);      \
  }

#define HOSTRPC_STAMP_FETCH(TYPE, NAME)                                   \
  DEVICE TYPE atomic_##NAME##_relaxed(volatile _Atomic(TYPE) * addr,      \
                                      TYPE value)                         \
  {                                                                       \
    return __opencl_atomic_##NAME(addr, value, __ATOMIC_RELAXED,          \
                                  __OPENCL_MEMORY_SCOPE_ALL_SVM_DEVICES); \
  }

#define HOSTRPC_STAMP_FETCH_OPS(TYPE)                                 \
  HOSTRPC_STAMP_FETCH(TYPE, fetch_add)                                \
  HOSTRPC_STAMP_FETCH(TYPE, fetch_and)                                \
  HOSTRPC_STAMP_FETCH(TYPE, fetch_or)                                 \
  DEVICE bool atomic_compare_exchange_weak_relaxed(                   \
      volatile _Atomic(TYPE) * addr, TYPE expected, TYPE desired,     \
      TYPE * loaded)                                                  \
  {                                                                   \
    bool r = __opencl_atomic_compare_exchange_weak(                   \
        addr, &expected, desired, __ATOMIC_RELAXED, __ATOMIC_RELAXED, \
        __OPENCL_MEMORY_SCOPE_ALL_SVM_DEVICES);                       \
    *loaded = expected;                                               \
    return r;                                                         \
  }

// Cuda maps uint64_t onto unsigned long while mangling, but it seems
// c++/ptx maps uint64_t onto unsigned long long
// Code calls:
//   platform::detail::atomic_load_relaxed(unsigned long long _Atomic const
//   volatile*)
// and this file implements:
//   platform::detail::atomic_load_relaxed(unsigned long _Atomic const
//   volatile*)
// despite both referring to uint64_t as their type
// hacking around here, but may be safer to use extern C symbols for all of
// these

HOSTRPC_STAMP_MEMORY(uint8_t)
HOSTRPC_STAMP_MEMORY(uint16_t)
HOSTRPC_STAMP_MEMORY(uint32_t)
HOSTRPC_STAMP_MEMORY(uint64_t)
HOSTRPC_STAMP_MEMORY(unsigned long long)

HOSTRPC_STAMP_FETCH_OPS(uint32_t)
HOSTRPC_STAMP_FETCH_OPS(uint64_t)
HOSTRPC_STAMP_FETCH_OPS(unsigned long long)

#undef HOSTRPC_STAMP_MEMORY
#undef HOSTRPC_STAMP_FETCH
#undef HOSTRPC_STAMP_FETCH_OPS

}  // namespace detail

}  // namespace platform
