#include <hip/hip_runtime.h>

// Intent is to use the cuda calls initially, then transform to clang intrinsics
// and move into platform.hpp

#define DEVICE __device__

#define WARPSIZE 32
DEVICE uint32_t get_lane_id(void) { return threadIdx.x & (WARPSIZE - 1); }

namespace detail
{
static DEVICE uint32_t ballot()
{
#if CUDA_VERSION >= 9000
  return __activemask();
#else
  return __ballot(1);
#endif
}

DEVICE uint32_t get_master_lane_id(void)
{
  uint32_t activemask = ballot();

  uint32_t lowest_active = __builtin_ffs(activemask) - 1;
  uint32_t lane_id = get_lane_id();

  return lane_id == lowest_active;

  // TODO: openmp deviceRTL uses:
  // return (blockDim.x - 1) & ~(WARPSIZE - 1);
}

DEVICE int32_t __impl_shfl_down_sync(int32_t var, uint32_t laneDelta)
{
  return __shfl_down_sync(UINT32_MAX, var, laneDelta, WARPSIZE);
}

}  // namespace detail

DEVICE uint32_t broadcast_master(uint32_t x)
{
  uint32_t master_id = detail::get_master_lane_id();
  // __nvvm_shfl_sync_idx_i32(UINT32_MAX, x, master_id, 31)
#if CUDA_VERSION >= 9000
  // Use activemask?
  return __shfl_sync(UINT32_MAX, x, master_id);
#else
  // This may be UB if some lanes are inactive
  return __shfl(x, master_id);
#endif
}
