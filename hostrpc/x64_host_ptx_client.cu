#include "x64_host_ptx_client_cuda.hpp"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

namespace hostrpc
{
namespace cuda
{
void *allocate_gpu(size_t size)
{
  void *ptr;
  hipError_t rc = hipMalloc(&ptr, size);
  if (rc != hipSuccess)
    {
      return nullptr;
    }

  // this runs asychronously with the host
  // documentation is not clear on how to tell when it has finished
  rc = hipMemset(ptr, 0, size);
  if (rc != hipSuccess)
    {
      return nullptr;
    }

  return ptr;
}

void deallocate_gpu(void *ptr)
{
  hipError_t rc = hipFree(ptr);
  (void)rc;
}

void *allocate_shared(size_t size)
{
  // hipHostRegister may be a better choice as the memory can be more easily
  // aligned that way. should check hipDeviceAttributeHostRegisterSupported
  void *ptr;
  hipError_t rc = hipHostAlloc(&ptr, size, hipHostMallocMapped);
  if (rc != hipSuccess)
    {
      return nullptr;
    }

  rc = hipMemset(ptr, 0, size);
  if (rc != hipSuccess)
    {
      return nullptr;
    }

  return ptr;
}

void deallocate_shared(void *ptr)
{
  hipError_t rc = hipHostFree(ptr);
  (void)rc;
}

void *device_ptr_from_host_ptr(void *host)
{
  void *device;
  unsigned int flags = 0;
  hipError_t rc = hipHostGetDevicePointer(&device, &host, flags);
  if (rc != hipSuccess)
    {
      return nullptr;
    }
  return device;
}

}  // namespace cuda
}  // namespace hostrpc
