// https://forums.developer.nvidia.com/t/does-anybody-have-experience-on-cudahostregister-zero-copy-memory/22539/3

#include <stdio.h>

#define SIZE 10

#include <hip/hip_runtime.h>

// Kernel definition, see also section 4.2.3 of Nvidia Cuda Programming Guide

__global__ void vecAdd(float *A, float *B, float *C)
{
  // threadIdx.x is a built-in variable provided by CUDA at runtime

  int i = threadIdx.x;

  //	A[i] = 0;
  //	B[i] = i;
  C[i] = A[i] + B[i];
  printf("Kernel: A[%d]=%f, B[%d]=%f, C[%d]=%f\n", i, A[i], i, B[i], i, C[i]);
}

int main()
{
  int N = SIZE;

  //	round up the size of the array to be a multiple of the page size

  size_t memsize = ((SIZE * sizeof(float) + 4095) / 4096) * 4096;

  hipDeviceProp_t deviceProp;

  // Get properties and verify device 0 supports mapped memory

  hipGetDeviceProperties(&deviceProp, 0);

  if (!deviceProp.canMapHostMemory)
    {
      fprintf(stderr, "Device %d cannot map host memory!\n", 0);

      exit(EXIT_FAILURE);
    }

  // set the device flags for mapping host memory

  hipSetDeviceFlags(hipDeviceMapHost);

  float *A, *B, *C;

  float *devPtrA, *devPtrB, *devPtrC;

  //	use valloc instead of malloc

  A = (float *)valloc(memsize);

  B = (float *)valloc(memsize);

  C = (float *)valloc(memsize);

  hipHostRegister(A, memsize, hipHostRegisterMapped);

  hipHostRegister(B, memsize, hipHostRegisterMapped);

  hipHostRegister(C, memsize, hipHostRegisterMapped);

  for (int i = 0; i < SIZE; i++)
    {
      A[i] = B[i] = i;
    }

  hipHostGetDevicePointer((void **)&devPtrA, (void *)A, 0);
  hipHostGetDevicePointer((void **)&devPtrB, (void *)B, 0);
  hipHostGetDevicePointer((void **)&devPtrC, (void *)C, 0);

  vecAdd<<<1, N>>>(devPtrA, devPtrB, devPtrC);

  hipDeviceSynchronize();

  for (int i = 0; i < SIZE; i++) printf("C[%d]=%f\n", i, C[i]);

  hipHostUnregister(A);
  hipHostUnregister(B);
  hipHostUnregister(C);

  free(A);
  free(B);
  free(C);
}
